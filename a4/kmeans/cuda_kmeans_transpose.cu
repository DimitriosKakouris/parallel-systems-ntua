#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "kmeans.h"
#include "alloc.h"
#include "error.h"

#ifdef __HIPCC__
inline void checkCuda(hipError_t e) {
    if (e != hipSuccess) {
        // hipGetErrorString() isn't always very helpful. Look up the error
        // number in the hipError_t enum in hip/driver_types.h in the CUDA includes
        // directory for a better explanation.
        error("CUDA Error %d: %s\n", e, hipGetErrorString(e));
    }
}

inline void checkLastCudaError() {
    checkCuda(hipGetLastError());
}
#endif

__device__ int get_tid(){

	int globalID = threadIdx.x + blockDim.x * blockIdx.x;
	return globalID; /* TODO: copy me from naive version... */
}

/* square of Euclid distance between two multi-dimensional points using column-base format */
__host__ __device__ inline static
double euclid_dist_2_transpose(int numCoords,
                    int    numObjs,
                    int    numClusters,
                    double *objects,     // [numCoords][numObjs]
                    double *clusters,    // [numCoords][numClusters]
                    int    objectId,
                    int    clusterId)
{
    int i;
    double ans=0.0;

	/* TODO: Calculate the euclid_dist of elem=objectId of objects from elem=clusterId from clusters, but for column-base format!!! */
    for (i=0; i<numCoords; i++){
        ans += (objects[i*numObjs + objectId] - clusters[i*numClusters + clusterId]) * (objects[i*numObjs + objectId] - clusters[i*numClusters + clusterId]);
     }
    return(ans);
}

__global__ static
void find_nearest_cluster(int numCoords,
                          int numObjs,
                          int numClusters,
                          double *objects,           //  [numCoords][numObjs]
                          double *deviceClusters,    //  [numCoords][numClusters]
                          int *membership,          //  [numObjs]
                          double *devdelta)
{
	/* TODO: copy me from naive version... */
    /* Get the global ID of the thread. */
    int tid = get_tid(); 

	/* TODO: Maybe something is missing here... should all threads run this? */
    if (tid<numObjs) {
        int   index, i;
        double dist, min_dist;

        /* find the cluster id that has min distance to object */
        index = 0;
        /* TODO: call min_dist = euclid_dist_2(...) with correct objectId/clusterId */
        min_dist = euclid_dist_2_transpose(numCoords,numObjs,numClusters,objects,deviceClusters,tid,index);

        for (i=1; i<numClusters; i++) {
            /* TODO: call dist = euclid_dist_2(...) with correct objectId/clusterId */
            dist = euclid_dist_2_transpose(numCoords,numObjs,numClusters,objects,deviceClusters,tid, i);
 
            /* no need square root */
            if (dist < min_dist) { /* find the min and its array index */
                min_dist = dist;
                index = i;
            }
        }

        if (membership[tid] != index) {
        	/* TODO: Maybe something is missing here... is this write safe? */
            atomicAdd(devdelta,1.0);
            /*(*devdelta)+= 1.0;*/
        }

        /* assign the deviceMembership to object objectId */
        membership[tid] = index;
    }
}

//
//  ----------------------------------------
//  DATA LAYOUT
//
//  objects         [numObjs][numCoords]
//  clusters        [numClusters][numCoords]
//  dimObjects      [numCoords][numObjs]
//  dimClusters     [numCoords][numClusters]
//  newClusters     [numCoords][numClusters]
//  deviceObjects   [numCoords][numObjs]
//  deviceClusters  [numCoords][numClusters]
//  ----------------------------------------
//
/* return an array of cluster centers of size [numClusters][numCoords]       */            
void kmeans_gpu(double *objects,      /* in: [numObjs][numCoords] */
		        int     numCoords,    /* no. features */
		        int     numObjs,      /* no. objects */
		        int     numClusters,  /* no. clusters */
		        double   threshold,    /* % objects change membership */
		        long    loop_threshold,   /* maximum number of iterations */
		        int    *membership,   /* out: [numObjs] */
				double *clusters,   /* out: [numClusters][numCoords] */
				int blockSize)  
{
    double timing = wtime(), timing_internal, timer_min = 1e42, timer_max = 0; 
    // double tmp_timer;
    // double CG_timer=0, GC_timer=0, G_timer=0, C_timer=0;
	// int    loop_iterations = 0; 
    int      i, j, index, loop=0;
    int     *newClusterSize; /* [numClusters]: no. objects assigned in each
                                new cluster */
    double  delta = 0, *dev_delta_ptr;          /* % of objects change their clusters */
    
    /* TODO: Transpose dims */
    double  **dimObjects = (double **)calloc_2d(numCoords, numObjs, sizeof(double));//calloc_2d(...) -> [numCoords][numObjs]
    double  **dimClusters = (double **)calloc_2d(numCoords, numClusters, sizeof(double));  //calloc_2d(...) -> [numCoords][numClusters]
    double  **newClusters = (double **)calloc_2d(numCoords, numClusters, sizeof(double));  //calloc_2d(...) -> [numCoords][numClusters]
    
    double *deviceObjects;
    double *deviceClusters;
    int *deviceMembership;

    printf("\n|-----------Transpose GPU Kmeans------------|\n\n");
    
    //  TODO: Copy objects given in [numObjs][numCoords] layout to new
   
	
//     for (j = 0; j < numObjs; j++) {
//         for (i = 0; i < numCoords; i++) {
//         dimObjects[i][j] = objects[j * numCoords + i];
//     }
// }
  for (i = 0; i < numCoords; i++) {
        for (j = 0; j < numObjs; j++) {
            dimObjects[i][j] = objects[j*numCoords + i];
        }
    }
	
    /* pick first numClusters elements of objects[] as initial cluster centers*/
    for (i = 0; i < numCoords; i++) {
        for (j = 0; j < numClusters; j++) {
            dimClusters[i][j] = dimObjects[i][j];
        }
    }

    
	
    /* initialize membership[] */
    for (i=0; i<numObjs; i++) membership[i] = -1;

    /* need to initialize newClusterSize and newClusters[0] to all 0 */
    newClusterSize = (int*) calloc(numClusters, sizeof(int));
    assert(newClusterSize != NULL); 
    
    timing = wtime() - timing;
    printf("t_alloc: %lf ms\n\n", 1000*timing);
    timing = wtime(); 

    const unsigned int numThreadsPerClusterBlock = (numObjs > blockSize)? blockSize: numObjs;
    const unsigned int numClusterBlocks = (numObjs + numThreadsPerClusterBlock - 1) / numThreadsPerClusterBlock; /* TODO: Calculate Grid size, e.g. number of blocks. */
    const unsigned int clusterBlockSharedDataSize = 0;
       
    checkCuda(hipMalloc(&deviceObjects, numObjs*numCoords*sizeof(double)));
    checkCuda(hipMalloc(&deviceClusters, numClusters*numCoords*sizeof(double)));
    checkCuda(hipMalloc(&deviceMembership, numObjs*sizeof(int)));
    checkCuda(hipMalloc(&dev_delta_ptr, sizeof(double)));
    timing = wtime() - timing;
    printf("t_alloc_gpu: %lf ms\n\n", 1000*timing);
    timing = wtime(); 
    
    checkCuda(hipMemcpy(deviceObjects, dimObjects[0],
              numObjs*numCoords*sizeof(double), hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(deviceMembership, membership,
              numObjs*sizeof(int), hipMemcpyHostToDevice));
    timing = wtime() - timing;
    printf("t_get_gpu: %lf ms\n\n", 1000*timing);
    timing = wtime();   


    #ifdef TIMING_ANALYSIS
    double gpu_time, gpu_cpu_time, cpu_time, cpu_gpu_time;
    double gpu_time_arr[10];
    double gpu_time_total = 0.0, gpu_cpu_time_total = 0.0, cpu_time_total = 0.0, cpu_gpu_time_total = 0.0;
    double gpu_time_min = __DBL_MAX__, gpu_cpu_time_min = __DBL_MAX__, cpu_time_min = __DBL_MAX__, cpu_gpu_time_min = __DBL_MAX__;
    double gpu_time_max = 0.0, gpu_cpu_time_max = 0.0, cpu_time_max = 0.0, cpu_gpu_time_max = 0.0;
    double time_start, time_end;
    #endif  
    
    do {
    	timing_internal = wtime();


        #ifdef TIMING_ANALYSIS
        time_start = wtime();
        #endif

		/* GPU part: calculate new memberships */
		        
        //tmp_timer=wtime();
        /* TODO: Copy clusters to deviceClusters */
        checkCuda(hipMemcpy(deviceClusters,dimClusters[0],numClusters*numCoords*sizeof(double), hipMemcpyHostToDevice));
        
        checkCuda(hipMemset(dev_delta_ptr, 0, sizeof(double))); 

        #ifdef TIMING_ANALYSIS
        time_end = wtime();
        cpu_gpu_time = time_end - time_start;
        time_start = wtime();
        #endif   


        //CG_timer += wtime()-tmp_timer;
        //tmp_timer=wtime();
		//printf("Launching find_nearest_cluster Kernel with grid_size = %d, block_size = %d, shared_mem = %d KB\n", numClusterBlocks, numThreadsPerClusterBlock, clusterBlockSharedDataSize/1000);
        find_nearest_cluster<<< numClusterBlocks, numThreadsPerClusterBlock, clusterBlockSharedDataSize >>>(numCoords, numObjs, numClusters,deviceObjects, deviceClusters, deviceMembership, dev_delta_ptr);

        hipDeviceSynchronize(); checkLastCudaError();

        #ifdef TIMING_ANALYSIS
        time_end = wtime();
        gpu_time = time_end - time_start;
        time_start = wtime();
        #endif

        //G_timer += wtime() - timing_internal;
		//printf("Kernels complete for itter %d, updating data in CPU\n", loop);
		
        //tmp_timer=wtime();
		/* TODO: Copy deviceMembership to membership*/
        checkCuda(hipMemcpy(membership,deviceMembership,numObjs*sizeof(int), hipMemcpyDeviceToHost));
    
    	/* TODO: Copy dev_delta_ptr to &delta*/
        checkCuda(hipMemcpy(&delta,dev_delta_ptr, sizeof(double),hipMemcpyDeviceToHost));
        //GC_timer += wtime()-tmp_timer;
        //tmp_timer=wtime();

        #ifdef TIMING_ANALYSIS
        time_end = wtime();
        gpu_cpu_time = time_end - time_start;
        time_start = wtime();
        #endif

		/* CPU part: Update cluster centers*/
  		
        for (i=0; i<numObjs; i++) {
            /* find the array index of nestest cluster center */
            index = membership[i];
			
            /* update new cluster centers : sum of objects located within */
            newClusterSize[index]++;
            for (j=0; j<numCoords; j++)
                newClusters[j][index] += objects[i*numCoords + j];
        }
 
        /* average the sum and replace old cluster centers with newClusters */
        for (i=0; i<numClusters; i++) {
            for (j=0; j<numCoords; j++) {
                if (newClusterSize[i] > 0)
                    dimClusters[j][i] = newClusters[j][i] / newClusterSize[i];
                newClusters[j][i] = 0.0;   /* set back to 0 */
            }
            newClusterSize[i] = 0;   /* set back to 0 */
        }

        //C_timer += wtime() - tmp_timer;

        

        delta /= numObjs;
       	//printf("delta is %f - ", delta);
        loop++; 




        #ifdef TIMING_ANALYSIS
        time_end = wtime();
        cpu_time = time_end - time_start;
        gpu_time_arr[loop] = gpu_time;

        gpu_time_total += gpu_time;
        gpu_cpu_time_total += gpu_cpu_time;
        cpu_time_total += cpu_time;
        cpu_gpu_time_total += cpu_gpu_time;
        if (gpu_time < gpu_time_min) gpu_time_min = gpu_time;
        if (gpu_time > gpu_time_max) gpu_time_max = gpu_time;
        if (gpu_cpu_time < gpu_cpu_time_min) gpu_cpu_time_min = gpu_cpu_time;
        if (gpu_cpu_time > gpu_cpu_time_max) gpu_cpu_time_max = gpu_cpu_time;
        if (cpu_time < cpu_time_min) cpu_time_min = cpu_time;
        if (cpu_time > cpu_time_max) cpu_time_max = cpu_time;
        if (cpu_gpu_time < cpu_gpu_time_min) cpu_gpu_time_min = cpu_gpu_time;
        if (cpu_gpu_time > cpu_gpu_time_max) cpu_gpu_time_max = cpu_gpu_time;
        #endif


        //printf("completed loop %d\n", loop);
		timing_internal = wtime() - timing_internal; 
		if ( timing_internal < timer_min) timer_min = timing_internal; 
		if ( timing_internal > timer_max) timer_max = timing_internal; 
	} while (delta > threshold && loop < loop_threshold);
    
    /*TODO: Update clusters using dimClusters. Be carefull of layout!!! clusters[numClusters][numCoords] vs dimClusters[numCoords][numClusters] */ 
// 	for (int i = 0; i < numClusters; i++) {
//     for (int j = 0; j < numCoords; j++) {
//         clusters[i*numCoords + j] = dimClusters[j][i];
//     }
// }
 for (i = 0; i < numCoords; i++) {
        for (j = 0; j < numClusters; j++) {
            clusters[j*numCoords + i] = dimClusters[i][j];
        }
    }
	
    timing = wtime() - timing;
    printf("nloops = %d  : total = %lf ms\n\t-> t_loop_avg = %lf ms\n\t-> t_loop_min = %lf ms\n\t-> t_loop_max = %lf ms\n\n|-------------------------------------------|\n", 
    	loop, 1000*timing, 1000*timing/loop, 1000*timer_min, 1000*timer_max);

    // printf("t_GPU->CPU = %lf ms\n", 1000*GC_timer); 
   	// printf("t_CPU->GPU = %lf ms\n", 1000*CG_timer); 
   	// printf("t_GPU = %lf ms\n", 1000*G_timer); 
   	// printf("t_CPU = %lf ms\n", 1000*C_timer); 

      // print timing information (avg, min, max) in each line
    #ifdef TIMING_ANALYSIS
    printf("GPU time: %lf ms\n\t-> t_loop_avg = %lf ms\n\t-> t_loop_min = %lf ms\n\t-> t_loop_max = %lf ms\n\n", 
    	1000*gpu_time_total, 1000*gpu_time_total/loop, 1000*gpu_time_min, 1000*gpu_time_max);
    printf("GPU-CPU time: %lf ms\n\t-> t_loop_avg = %lf ms\n\t-> t_loop_min = %lf ms\n\t-> t_loop_max = %lf ms\n\n",
        1000*gpu_cpu_time_total, 1000*gpu_cpu_time_total/loop, 1000*gpu_cpu_time_min, 1000*gpu_cpu_time_max);
    printf("CPU time: %lf ms\n\t-> t_loop_avg = %lf ms\n\t-> t_loop_min = %lf ms\n\t-> t_loop_max = %lf ms\n\n",
        1000*cpu_time_total, 1000*cpu_time_total/loop, 1000*cpu_time_min, 1000*cpu_time_max);
    printf("CPU-GPU time: %lf ms\n\t-> t_loop_avg = %lf ms\n\t-> t_loop_min = %lf ms\n\t-> t_loop_max = %lf ms\n\n",
        1000*cpu_gpu_time_total, 1000*cpu_gpu_time_total/loop, 1000*cpu_gpu_time_min, 1000*cpu_gpu_time_max);
    #endif


	char outfile_name[1024] = {0}; 
	sprintf(outfile_name, "Execution_logs/silver1-V100_Sz-%lu_Coo-%d_Cl-%d.csv", numObjs*numCoords*sizeof(double)/(1024*1024), numCoords, numClusters);
	FILE* fp = fopen(outfile_name, "a+");
	if(!fp) error("Filename %s did not open succesfully, no logging performed\n", outfile_name); 

    #ifndef TIMING_ANALYSIS
	fprintf(fp, "%s,%d,%lf,%lf,%lf,%lf\n", "Transpose", blockSize, timing/loop, timer_min, timer_max,timing);
    #endif
    
    #ifdef TIMING_ANALYSIS
    fprintf(fp, "%s,%d,%lf,%lf,%lf,%lf,%lf,%lf,%lf,%lf \n", "Transpose", blockSize, timing/loop, timer_min, timer_max,timing,1000*gpu_cpu_time_total, 1000*cpu_gpu_time_total, 1000*gpu_time_total,1000*cpu_time_total);
    #endif

	fclose(fp); 
	
    checkCuda(hipFree(deviceObjects));
    checkCuda(hipFree(deviceClusters));
    checkCuda(hipFree(deviceMembership));

    free(dimObjects[0]);
    free(dimObjects);
    free(dimClusters[0]);
    free(dimClusters);
    free(newClusters[0]);
    free(newClusters);
    free(newClusterSize);

    return;
}

